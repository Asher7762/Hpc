#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <chrono>

#define BLOCK_SIZE 256
#define CUDA_CORES 768

__global__ void vectorAddShared(int* A, int* B, int* C, int n) {
    __shared__ int s_A[BLOCK_SIZE];
    __shared__ int s_B[BLOCK_SIZE];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        s_A[threadIdx.x] = A[idx];
        s_B[threadIdx.x] = B[idx];
        __syncthreads();

        C[idx] = s_A[threadIdx.x] + s_B[threadIdx.x];
    }
}

void vectorAddCPU(int* A, int* B, int* C, int n) {
    for (int i = 0; i < n; ++i)
        C[i] = A[i] + B[i];
}

int main() {
    int sizes[5] = {100000, 5000000, 8000000, 10000000, 30000000};
    printf("\nRohit Chauhan BE A 41008)\n");
    printf("\nVector Addition Benchmark (Shared Memory)\n");
    printf("---------------------------------------------------------------\n");
    printf("| Vector Size | CPU Time(s) | GPU Time(s) | Speedup | Efficiency |\n");
    printf("---------------------------------------------------------------\n");

    for (int i = 0; i < 5; i++) {
        int N = sizes[i];
        int *h_A = (int*)malloc(N * sizeof(int));
        int *h_B = (int*)malloc(N * sizeof(int));
        int *h_C_CPU = (int*)malloc(N * sizeof(int));
        int *h_C_GPU = (int*)malloc(N * sizeof(int));

        for (int j = 0; j < N; ++j) {
            h_A[j] = rand() % 100;
            h_B[j] = rand() % 100;
        }

        // CPU time
        auto start_cpu = std::chrono::high_resolution_clock::now();
        vectorAddCPU(h_A, h_B, h_C_CPU, N);
        auto end_cpu = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> cpu_duration = end_cpu - start_cpu;
        double cpu_time = cpu_duration.count();

        // Allocate device memory
        int *d_A, *d_B, *d_C;
        hipMalloc((void**)&d_A, N * sizeof(int));
        hipMalloc((void**)&d_B, N * sizeof(int));
        hipMalloc((void**)&d_C, N * sizeof(int));

        hipMemcpy(d_A, h_A, N * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, N * sizeof(int), hipMemcpyHostToDevice);

        // GPU time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        int gridSize = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

        hipEventRecord(start);
        vectorAddShared<<<gridSize, BLOCK_SIZE>>>(d_A, d_B, d_C, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float gpu_time_ms;
        hipEventElapsedTime(&gpu_time_ms, start, stop);
        double gpu_time = gpu_time_ms / 1000.0;

        hipMemcpy(h_C_GPU, d_C, N * sizeof(int), hipMemcpyDeviceToHost);

        double speedup = cpu_time / gpu_time;
        double efficiency = speedup / CUDA_CORES;

        printf("| %11d | %11.6f | %11.6f | %7.2f | %9.6f |\n",
               N, cpu_time, gpu_time, speedup, efficiency);

        hipFree(d_A); hipFree(d_B); hipFree(d_C);
        free(h_A); free(h_B); free(h_C_CPU); free(h_C_GPU);
        hipEventDestroy(start); hipEventDestroy(stop);
    }

    printf("---------------------------------------------------------------\n");
    return 0;
}

